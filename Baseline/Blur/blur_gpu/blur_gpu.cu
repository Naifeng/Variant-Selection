// Compile it with:
// nvcc blur_gpu.cu -o blur_gpu
// Run it with:
// CUDA_VISIBLE_DEVICES=1 ./blur_gpu



#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <math.h>
#include <stdio.h>
#include <assert.h>
#include <fstream>
#include <time.h>


__global__ void convolutionGPU(
float *d_Result,
float *d_Data,
int dataW,
int dataH )
{

// global mem address for this thread
const int gLoc = threadIdx.x +
blockIdx.x * blockDim.x +
threadIdx.y * dataW +
blockIdx.y * blockDim.y * dataW;

float sum = 0;
float value = 0;

int KERNEL_RADIUS = 3;


for (int i = -KERNEL_RADIUS; i <= KERNEL_RADIUS; i++) // row wise
for (int j = -KERNEL_RADIUS; j <= KERNEL_RADIUS; j++) // col wise
{
// check row first
if (blockIdx.x == 0 && (threadIdx.x + i) < 0) // left apron
value = 0;
else if ( blockIdx.x == (gridDim.x - 1) &&
(threadIdx.x + i) > blockDim.x-1 ) // right apron
value = 0;
else
{
// check col next
if (blockIdx.y == 0 && (threadIdx.y + j) < 0) // top apron
value = 0;
else if ( blockIdx.y == (gridDim.y - 1) &&
(threadIdx.y + j) > blockDim.y-1 ) // bottom apron
value = 0;
else // safe case
value = d_Data[gLoc + i + j * dataW];
}
sum += value * 0.5 * 0.5;
}
d_Result[gLoc] = sum;
}


void image_convolution(float * input,float* output, int img_height, int img_width, const int r, float & gpu_elapsed_time_ms)
{


    // initialize kernel here
    int kernel_height = r;
    int kernel_width = r;

    float *kernel;
    kernel = new float[r*r];

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);


    hipEventRecord(start, 0);

    for (int i = 0; i < r*r; i++){
        kernel[i] = rand() % 10 + 1;
    }


    float * mask = new float[kernel_height*kernel_width];
    for (int i = 0; i < kernel_height*kernel_width; i++)
    {
        mask[i] = kernel[i];
    }



    float * d_input, * d_output, * d_kernel;
    hipMalloc(&d_input, img_width*img_height*sizeof(float));
    hipMalloc(&d_output, img_width*img_height*sizeof(float));
    hipMalloc(&d_kernel, kernel_height*kernel_width*sizeof(float));

    hipMemcpy(d_input, input, img_width*img_height*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_kernel, mask, kernel_height*kernel_width*sizeof(float), hipMemcpyHostToDevice);
    dim3 blocksize(16,16);
    dim3 gridsize;
    gridsize.x=(img_width+blocksize.x-1)/blocksize.x;
    gridsize.y=(img_height+blocksize.y-1)/blocksize.y;


    convolutionGPU<<<gridsize,blocksize>>>(d_output,d_input,img_width,img_height);
    
    hipMemcpy(output, d_output, img_width*img_height*sizeof(float), hipMemcpyDeviceToHost);


    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&gpu_elapsed_time_ms, start, stop);
}

int main(){
    
    // number of instances of data generated
    int NUM = 5;
    float total_time = 0;

    std::ofstream ofile;

    // change here to customize output filename
    ofile.open("naive_blur_gpu.csv");

    for (int iterator = 0; iterator < NUM; iterator++) {


        // currently have to manually change the input size
        float *in, *out;
        int m = 16384;
        int n = 16384;
        int is = n * m;

        int r = 3;

        in = new float[is];
        out = new float[is];


        for (int i = 0; i < m * n; i++)
            in[i] = rand() % 1024 + 1;

        
        float time;

        image_convolution(in, out, n, m, r, time);
        
        total_time += time;


    }

    std::cout << total_time / (NUM*1000) << std::endl;

    ofile.close();
    return 0;
}